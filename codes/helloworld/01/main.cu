
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void MyCudaPrint();

__global__ void MyCudaPrint() {
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    printf("[ %d ] HelloWorld\n", idx);	
}

int main(int argc, char* argv[])
{
    MyCudaPrint <<<1, 1 >>> ();

    hipDeviceSynchronize();

    return 0;
}