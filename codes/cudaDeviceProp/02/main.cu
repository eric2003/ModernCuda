#include <cstdio>
#include <hip/hip_runtime.h>

int main(int argc, char *argv[])
{
    int num_gpus = 0;    
    hipGetDeviceCount( &num_gpus );

    for ( int iGpu = 0; iGpu < num_gpus; ++ iGpu )
    {
       hipDeviceProp_t prop;
       hipGetDeviceProperties( &prop, iGpu );

       std::printf( " --- General Information for device %d ---\n", iGpu );
       std::printf( "Device Name: %s\n", prop.name );
       std::printf( "Compute capability: %d.%d\n", prop.major, prop.minor );
       std::printf( "Clock rate: %d\n", prop.clockRate );
       std::printf( "Device copy overlap: " );
       
       if ( prop.deviceOverlap )
       {
            std::printf( "Enabled\n" );
       }
       else
       {
            std::printf( "Disabled\n" );
       }
       std::printf( "Kernel execition timeout : " );


       if ( prop.kernelExecTimeoutEnabled )
       {
            std::printf( "Enabled\n" );
       }
       else
       {
            std::printf( "Disabled\n" );
       }
    
       std::printf( " --- Memory Information for device %d ---\n", iGpu );
       std::printf( "Total global mem: %zd\n", prop.totalGlobalMem );
       std::printf( "Total constant Mem: %zd\n", prop.totalConstMem );
       std::printf( "Max mem pitch: %zd\n", prop.memPitch );
       std::printf( "Texture Alignment: %zd\n", prop.textureAlignment );
       std::printf( " --- MP Information for device %d ---\n", iGpu );
       std::printf( "Multiprocessor count: %d\n",prop.multiProcessorCount );
       std::printf( "Shared mem per mp: %zd\n", prop.sharedMemPerBlock );
       std::printf( "Registers per mp: %d\n", prop.regsPerBlock );
       std::printf( "Threads in warp: %d\n", prop.warpSize );
       std::printf( "Max threads per block: %d\n",prop.maxThreadsPerBlock );
       std::printf( "Max thread dimensions[0 - 2]: (%d, %d, %d)\n", prop.maxThreadsDim[0], prop.maxThreadsDim[1],prop.maxThreadsDim[2] );
       std::printf( "Max grid dimensions[0 - 2]: (%d, %d, %d)\n",prop.maxGridSize[0], prop.maxGridSize[1],prop.maxGridSize[2] );
       std::printf( "\n" );
    }
    
    return 0;
}
