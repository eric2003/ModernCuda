#include <cstdio>
#include <hip/hip_runtime.h>

__global__ void printDimInfo()
{
    int blockId = blockIdx.x + blockIdx.y * gridDim.x
        + gridDim.x * gridDim.y * blockIdx.z;
    int threadId = blockId * (blockDim.x * blockDim.y * blockDim.z)
        + (threadIdx.z * (blockDim.x * blockDim.y))
        + (threadIdx.y * blockDim.x) + threadIdx.x;

    if ( threadId == 0 )
    {
        std::printf("block Dim =[%d,%d,%d]\n", blockDim.x, blockDim.y, blockDim.z);
        std::printf("grid  Dim =[%d,%d,%d]\n", gridDim.x, gridDim.y, gridDim.z);
        std::printf("\n");
    }
}

int main(int argc, char *argv[])
{
    dim3 dimGrid(4, 3, 2);
    dim3 dimBlock(2, 4, 6);
    printDimInfo<<<1, 10 >>>();
    printDimInfo<<<dimGrid, dimBlock >>>();

    hipDeviceSynchronize();
    
    return 0;
}
