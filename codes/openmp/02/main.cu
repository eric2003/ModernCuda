
#include <hip/hip_runtime.h>
#include <omp.h>
#include <cstdio>

int main(int argc, char* argv[])
{
    int num_gpus = 0;
    
    std::printf("%s Starting...\n\n", argv[0]);
    
    hipGetDeviceCount(&num_gpus);
    
    if ( num_gpus < 1 ) {
        std::printf("no CUDA capable devices were detected\n");
        return 1;
    }
    
    /////////////////////////////////////////////////////////////////
    // display CPU and GPU configuration
    //
    std::printf("number of host CPUs:\t%d\n", omp_get_num_procs());
    std::printf("number of CUDA devices:\t%d\n", num_gpus);
    
    for ( int i = 0; i < num_gpus; ++ i )
	{
        hipDeviceProp_t dprop;
        hipGetDeviceProperties(&dprop, i);
        std::printf("   %d: %s\n", i, dprop.name);
    }

    std::printf("---------------------------\n");
	
#pragma omp parallel
    {
        unsigned int cpu_thread_id = omp_get_thread_num();
        unsigned int num_cpu_threads = omp_get_num_threads();
        std::printf("CPU thread %d (of %d)\n", cpu_thread_id, num_cpu_threads);
    }
    return 0;
}
