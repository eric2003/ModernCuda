
#include <hip/hip_runtime.h>
#include <omp.h>
#include <cstdio>

int main(int argc, char* argv[])
{
    std::printf("number of host CPUs:\t%d\n", omp_get_num_procs());
    int nThreads = 2;
    std::printf("nThreads=%d\n", nThreads);
    omp_set_num_threads(nThreads);
#pragma omp parallel
    {
        unsigned int cpu_thread_id = omp_get_thread_num();
        unsigned int num_cpu_threads = omp_get_num_threads();
        std::printf("CPU thread %d (of %d)\n", cpu_thread_id, num_cpu_threads);
    }
    return 0;
}
