#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <hip/hip_runtime.h>
#include <ctime>
#include <chrono>
#include "TimeSpan.h"

__global__ void vectorAdd( const float *A, const float *B, float *C, int numElements )
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    
    if ( i < numElements )
    {
        C[i] = A[i] + B[i] + 0.0f;
    }
}

int main(int argc, char* argv[])
{
    //int numElements = 50000;
    int numElements = 50000000;
    size_t size = numElements * sizeof(float);
    std::printf("[Vector addition of %d elements]\n", numElements);
    TimeSpan ts;
    clock_t clock_start;
    clock_t clock_end;  
    std::chrono::time_point<std::chrono::system_clock> c11_start, c11_end;
    using clock_type = std::chrono::time_point<std::chrono::system_clock>;
    clock_type time_now, time_old;
    typedef std::chrono::steady_clock Clock;
    Clock::time_point clock_t1, clock_t2;
    float elapsedTime = 0.0;
    hipEvent_t event_start, event_stop;
    hipEventCreate(&event_start);
    hipEventCreate(&event_stop);
    hipEventRecord(event_start, 0);
    clock_start = clock();
    c11_start = std::chrono::system_clock::now();
    time_old = std::chrono::system_clock::now();
    clock_t1 = std::chrono::steady_clock::now();
    ts.Start();
    
    float *h_A = (float *)std::malloc(size);
    float *h_B = (float *)std::malloc(size);
    float *h_C = (float *)std::malloc(size);
    
    for ( int i = 0; i < numElements; ++i )
    {
        h_A[i] = rand() / (float)RAND_MAX;
        h_B[i] = rand() / (float)RAND_MAX;
    }
    
    float *d_A = NULL;
    float *d_B = NULL;
    float *d_C = NULL;
    hipMalloc((void **)&d_A, size);
    hipMalloc((void **)&d_B, size);
    hipMalloc((void **)&d_C, size);
    
    std::printf("Copy input data from the host memory to the CUDA device\n");
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);
    
    // Launch the Vector Add CUDA Kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = ( numElements + threadsPerBlock - 1 ) / threadsPerBlock;
    std::printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid,
               threadsPerBlock);
    vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, numElements);
    std::printf("Copy output data from the CUDA device to the host memory\n");
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);
    
    // Verify that the result vector is correct
    for ( int i = 0; i < numElements; ++ i )
    {
        if ( std::fabs(h_A[i] + h_B[i] - h_C[i]) > 1e-5 )
        {
            fprintf(stderr, "Result verification failed at element %d!\n", i);
            std::exit(EXIT_FAILURE);
        }
    }
    
    std::printf("Test PASSED\n");
    
    hipEventRecord(event_stop, 0);
    hipEventSynchronize(event_stop);
    hipEventElapsedTime(&elapsedTime, event_start, event_stop);
    std::printf("cudaevent time = %lfms\n", elapsedTime);
    {
        hipDeviceSynchronize();
        clock_end= clock();
        double clock_diff_sec = ((double)(clock_end- clock_start) / CLOCKS_PER_SEC);
        std::printf("clock_ time: %lfms.\n", clock_diff_sec * 1000);    
    }
    {
        hipDeviceSynchronize();
        c11_end = std::chrono::system_clock::now();
        int elapsed_seconds = std::chrono::duration_cast<std::chrono::milliseconds>
                             (c11_end-c11_start).count();
        std::printf("chrono time: %dms.\n", elapsed_seconds);
    }
    {
        hipDeviceSynchronize();
        ts.ShowTimeSpan();
    }
    {
        hipDeviceSynchronize();
        time_now = std::chrono::system_clock::now();
        double elapsed_ms = std::chrono::duration_cast<std::chrono::milliseconds>
                             (time_now-time_old).count();
        std::printf("chrono time: %lfms.\n", elapsed_ms);
    }
    {
        hipDeviceSynchronize();
        time_now = std::chrono::system_clock::now();
        std::cout << "time_old.time_since_epoch().count() = " << time_old.time_since_epoch().count() << std::endl;
        std::cout << "time_now.time_since_epoch().count() = " << time_now.time_since_epoch().count() << std::endl;      
        std::cout << "(time_now - time_old).count() = " << (time_now - time_old).count() << std::endl;      
        std::cout << " time elapsed : " 
                  << std::chrono::duration_cast<std::chrono::milliseconds>(time_now - time_old).count()
                  << " milliseconds" << std::endl;      
    }
    {
        hipDeviceSynchronize();
        clock_t2 = std::chrono::steady_clock::now();
        std::cout << "clock_t1.time_since_epoch().count() = " << clock_t1.time_since_epoch().count() << std::endl;
        std::cout << "clock_t2.time_since_epoch().count() = " << clock_t2.time_since_epoch().count() << std::endl;
        std::chrono::steady_clock::duration time_span = clock_t2 - clock_t1;
        std::cout << "(clock_t2 - clock_t1).count() = " << (clock_t2 - clock_t1).count() << std::endl;      
        std::cout << " time elapsed : " 
                  << double(std::chrono::duration_cast<std::chrono::milliseconds>(time_span).count())
                  << " milliseconds" << std::endl;
    }   
    
    
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    
    std::free(h_A);
    std::free(h_B);
    std::free(h_C);
    
    std::printf("Done\n");
    return 0;
}
