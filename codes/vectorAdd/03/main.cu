#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <hip/hip_runtime.h>
#include <ctime>
#include <chrono>
#include "TimeSpan.h"

__global__ void vectorAdd( const float *A, const float *B, float *C, int numElements )
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    
    if ( i < numElements )
    {
        C[i] = A[i] + B[i] + 0.0f;
    }
}

void Check( const float *A, const float *B, const float *C, int numElements )
{
    // Verify that the result vector is correct
    for ( int i = 0; i < numElements; ++ i )
    {
        if ( std::fabs(A[i] + B[i] - C[i]) > 1e-5 )
        {
            fprintf(stderr, "Result verification failed at element %d!\n", i);
            std::exit(EXIT_FAILURE);
        }
    }
    std::printf("Test PASSED\n");
}

void vectorAddSerial( const float *A, const float *B, float *C, int numElements )
{
    // Verify that the result vector is correct
    for ( int i = 0; i < numElements; ++ i )
    {
        C[i] = A[i] + B[i] + 0.0f;
    }
}

void vectorAddSerial()
{
    int numElements = 50000000;
    size_t size = numElements * sizeof(float);
    std::printf("[Vector addition of %d elements]\n", numElements);
   
    float *h_A = (float *)std::malloc(size);
    float *h_B = (float *)std::malloc(size);
    float *h_C = (float *)std::malloc(size);
	
    
    for ( int i = 0; i < numElements; ++i )
    {
        h_A[i] = rand() / (float)RAND_MAX;
        h_B[i] = rand() / (float)RAND_MAX;
    }
	
    TimeSpan ts;
    ts.Start();
	
	vectorAddSerial( h_A, h_B, h_C, numElements );
	
	ts.ShowTimeSpan("vectorAddSerial");	
	
	Check( h_A, h_B, h_C, numElements );
	
    
    std::free(h_A);
    std::free(h_B);
    std::free(h_C);
    
    std::printf("Done\n");
}

int main(int argc, char* argv[])
{
	vectorAddSerial();
    //int numElements = 50000;
    int numElements = 50000000;
    size_t size = numElements * sizeof(float);
    std::printf("[Vector addition of %d elements]\n", numElements);
    
    float *h_A = (float *)std::malloc(size);
    float *h_B = (float *)std::malloc(size);
    float *h_C = (float *)std::malloc(size);
    
    for ( int i = 0; i < numElements; ++i )
    {
        h_A[i] = rand() / (float)RAND_MAX;
        h_B[i] = rand() / (float)RAND_MAX;
    }
    
    float *d_A = NULL;
    float *d_B = NULL;
    float *d_C = NULL;
    hipMalloc((void **)&d_A, size);
    hipMalloc((void **)&d_B, size);
    hipMalloc((void **)&d_C, size);
    
    std::printf("Copy input data from the host memory to the CUDA device\n");
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);
    
    // Launch the Vector Add CUDA Kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = ( numElements + threadsPerBlock - 1 ) / threadsPerBlock;
    std::printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid,
               threadsPerBlock);
    TimeSpan ts;
    ts.Start();
    vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, numElements);
    {
        hipDeviceSynchronize();
        ts.ShowTimeSpan("vectorAddCUDA");
    }

    std::printf("Copy output data from the CUDA device to the host memory\n");
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);
	
	Check( h_A, h_B, h_C, numElements );
    
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    
    std::free(h_A);
    std::free(h_B);
    std::free(h_C);
    
    std::printf("Done\n");
    return 0;
}
