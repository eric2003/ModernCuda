
#include <hip/hip_runtime.h>
#include <omp.h>
#include <cstdio>

__global__ void kernelAddConstant(int *g_a, const int b)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    g_a[ idx ] += b;
}

int CheckResult( int *data, const int n, const int b )
{
    for ( int i = 0; i < n; ++ i )
    {
        if ( data[i] != i + b ) return 0;
    }
    
    return 1;
}

int main(int argc, char* argv[])
{
    int num_gpus = 0;
    
    std::printf("%s Starting...\n\n", argv[0]);
    
    hipGetDeviceCount(&num_gpus);
    
    if ( num_gpus < 1 ) {
        std::printf("no CUDA capable devices were detected\n");
        return 1;
    }
    
    std::printf("number of host CPUs:\t%d\n", omp_get_num_procs());
    std::printf("number of CUDA devices:\t%d\n", num_gpus);
    
    for ( int i = 0; i < num_gpus; ++ i )
    {
        hipDeviceProp_t dprop;
        hipGetDeviceProperties(&dprop, i);
        std::printf("   %d: %s\n", i, dprop.name);
    }

    std::printf("---------------------------\n");

    unsigned int n = num_gpus * 8192;
    unsigned int nbytes = n * sizeof(int);

    int * a = (int *)std::malloc( nbytes );
    int b   = 3;    
    
    for ( unsigned int i = 0; i < n; ++ i )
    {
        a[i] = i;
    }
    
    omp_set_num_threads(2*num_gpus);
#pragma omp parallel
    {
        unsigned int cpu_thread_id = omp_get_thread_num();
        unsigned int num_cpu_threads = omp_get_num_threads();
        int gpu_id = -1;
        hipSetDevice(cpu_thread_id % num_gpus );
        hipGetDevice(&gpu_id);
        std::printf("CPU thread %d (of %d) uses CUDA device %d\n", cpu_thread_id,num_cpu_threads, gpu_id);
        int *d_a = 0;
        int *sub_a = a + cpu_thread_id * n / num_cpu_threads;
               
        unsigned int nbytes_per_kernel = nbytes / num_cpu_threads;
        dim3 gpu_threads(128);  // 128 threads per block
        dim3 gpu_blocks( n / ( gpu_threads.x * num_cpu_threads ) );
        std::printf("gpu_threads.x=%d gpu_threads.y=%d gpu_threads.z=%d\n", gpu_threads.x,gpu_threads.y, gpu_threads.z);
        std::printf("gpu_blocks.x=%d gpu_blocks.y=%d gpu_blocks.z=%d\n", gpu_blocks.x,gpu_blocks.y, gpu_blocks.z);
        
        hipMalloc( (void **)&d_a, nbytes_per_kernel );
        hipMemset( d_a, 0, nbytes_per_kernel );
        hipMemcpy( d_a, sub_a, nbytes_per_kernel, hipMemcpyHostToDevice );
        kernelAddConstant<<<gpu_blocks, gpu_threads>>>( d_a, b );
        
        hipMemcpy( sub_a, d_a, nbytes_per_kernel, hipMemcpyDeviceToHost );
        hipFree( d_a );
    }
    bool bResult = CheckResult(a, n, b);
    if ( bResult )
    {
        std::printf( "CHECK PASSED!\n" );
    }
    else
    {
        std::printf( "CHECK FAILED!\n" );
    }
    std::free(a);
    return 0;
}
