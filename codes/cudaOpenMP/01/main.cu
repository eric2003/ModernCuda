
#include <hip/hip_runtime.h>
#include <omp.h>
#include <cstdio>

int main(int argc, char* argv[])
{
    int num_gpus = 0;
    
    std::printf("%s Starting...\n\n", argv[0]);
    
    hipGetDeviceCount(&num_gpus);
    
    if ( num_gpus < 1 ) {
        std::printf("no CUDA capable devices were detected\n");
        return 1;
    }
    
    /////////////////////////////////////////////////////////////////
    // display CPU and GPU configuration
    //
    std::printf("number of host CPUs:\t%d\n", omp_get_num_procs());
    std::printf("number of CUDA devices:\t%d\n", num_gpus);
    
    for ( int i = 0; i < num_gpus; ++ i )
    {
        hipDeviceProp_t dprop;
        hipGetDeviceProperties(&dprop, i);
        std::printf("   %d: %s\n", i, dprop.name);
    }

    std::printf("---------------------------\n");
    
    omp_set_num_threads(2*num_gpus);
    
#pragma omp parallel
    {
        unsigned int cpu_thread_id = omp_get_thread_num();
        unsigned int num_cpu_threads = omp_get_num_threads();
        // set and check the CUDA device for this CPU thread
        int gpu_id = -1;
        hipSetDevice(cpu_thread_id % num_gpus );
        hipGetDevice(&gpu_id);
        std::printf("CPU thread %d (of %d) uses CUDA device %d\n", cpu_thread_id,num_cpu_threads, gpu_id);
    }
    return 0;
}
